#include "hip/hip_runtime.h"
/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 * Copyright (c) 2018 University of Maryland, College Park
 * Licensed under The Apache-2.0 License [see LICENSE for details]
 * \file multi_proposal.cc
 * \brief Proposal target layer
*/

#include "./multi_proposal-inl.h"
#include <set>
#include <math.h>
#include <unistd.h>
#include <dmlc/logging.h>
#include <dmlc/parameter.h>
#include <mxnet/operator.h>
#include <mshadow/tensor.h>
#include <mshadow/cuda/reduce.cuh>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <thrust/functional.h>
#include "./operator_common.h"
#include "./mshadow_op.h"
#include <time.h>
#include <stdlib.h> 

//============================
// Bounding Box Transform Utils
//============================
namespace mxnet {
namespace op {
namespace utils {

inline void BBoxTransformInv(float* boxes,
                             float* deltas,
                             float* im_info,
                             int num_images,
                             int anchors,
                             int heights,
                             int widths) {
  int num_anchors = anchors * heights * widths;
  //usleep(20000000);
  #pragma omp parallel for num_threads(8)
  for (int t = 0; t < num_images * num_anchors; ++t) {
    int b = t / num_anchors;
    int index = t % num_anchors;
    int a = index / (heights*widths);
    int mat = index % (heights*widths);
    int w = mat % widths; //width index
    int h = mat / widths; //height index
    float width = boxes[5*t + 2] - boxes[5*t] + 1.0;
    float height = boxes[5*t + 3] - boxes[5*t + 1] + 1.0;
    float ctr_x = boxes[5*t + 0] + 0.5 * (width - 1.0);
    float ctr_y = boxes[5*t + 1] + 0.5 * (height - 1.0);
    float dx = deltas[b*num_anchors*4 + a*4*widths*heights + h*widths + w];
    float dy = deltas[b*num_anchors*4 + (a*4 + 1)*widths*heights + h*widths + w];
    float dw = deltas[b*num_anchors*4 + (a*4 + 2)*widths*heights + h*widths + w];
    float dh = deltas[b*num_anchors*4 + (a*4 + 3)*widths*heights + h*widths + w];
    float pred_ctr_x = dx * width + ctr_x;
    float pred_ctr_y = dy * height + ctr_y;
    float pred_w = exp(dw) * width;
    float pred_h = exp(dh) * height;

    float pred_x1 = pred_ctr_x - 0.5 * (pred_w - 1.0);
    float pred_y1 = pred_ctr_y - 0.5 * (pred_h - 1.0);
    float pred_x2 = pred_ctr_x + 0.5 * (pred_w - 1.0);
    float pred_y2 = pred_ctr_y + 0.5 * (pred_h - 1.0);

    pred_x1 = std::max(std::min(pred_x1, im_info[3*b+1] - 1.0f), 0.0f);
    pred_y1 = std::max(std::min(pred_y1, im_info[3*b] - 1.0f), 0.0f);
    pred_x2 = std::max(std::min(pred_x2, im_info[3*b+1] - 1.0f), 0.0f);
    pred_y2 = std::max(std::min(pred_y2, im_info[3*b] - 1.0f), 0.0f);

    boxes[5*t] = pred_x1;
    boxes[5*t + 1] = pred_y1;
    boxes[5*t + 2] = pred_x2;
    boxes[5*t + 3] = pred_y2;
  }
}

// filter box by set confidence to zero
// * height or width < rpn_min_size
inline void FilterBox(float *dets,
                      int num_dets, float min_size) {
  #pragma omp parallel for num_threads(8)
  for (int i = 0; i < num_dets; ++i) {
    float iw = dets[5*i + 2] - dets[5*i] + 1.0f;
    float ih = dets[5*i + 3] - dets[5*i + 1] + 1.0f;
    if (iw < min_size || ih < min_size) {
      dets[5*i+0] -= min_size / 2;
      dets[5*i+1] -= min_size / 2;
      dets[5*i+2] += min_size / 2;
      dets[5*i+3] += min_size / 2;
      dets[5*i+4] = -1.0f;
    }
  }
}


inline void _MakeAnchor(float w,
                        float h,
                        float x_ctr,
                        float y_ctr,
                        std::vector<float> *out_anchors) {
  out_anchors->push_back(x_ctr - 0.5f * (w - 1.0f));
  out_anchors->push_back(y_ctr - 0.5f * (h - 1.0f));
  out_anchors->push_back(x_ctr + 0.5f * (w - 1.0f));
  out_anchors->push_back(y_ctr + 0.5f * (h - 1.0f));
}

inline void _Transform(float scale,
                       float ratio,
                       const std::vector<float>& base_anchor,
                       std::vector<float>  *out_anchors) {
  float w = base_anchor[2] - base_anchor[0] + 1.0f;
  float h = base_anchor[3] - base_anchor[1] + 1.0f;
  float x_ctr = base_anchor[0] + 0.5 * (w - 1.0f);
  float y_ctr = base_anchor[1] + 0.5 * (h - 1.0f);
  float size = w * h;
  float size_ratios = std::floor(size / ratio);
  float new_w = std::floor(std::sqrt(size_ratios) + 0.5f) * scale;
  float new_h = std::floor((new_w / scale * ratio) + 0.5f) * scale;

  _MakeAnchor(new_w, new_h, x_ctr,
             y_ctr, out_anchors);
}

// out_anchors must have shape (n, 5), where n is ratios.size() * scales.size()
inline void GenerateAnchors(const std::vector<float>& base_anchor,
                            const nnvm::Tuple<float>& ratios,
                            const nnvm::Tuple<float>& scales,
                            std::vector<float> *out_anchors) {

  for (size_t j = 0; j < ratios.ndim(); ++j) {
    for (size_t k = 0; k < scales.ndim(); ++k) {
      _Transform(scales[k], ratios[j], base_anchor, out_anchors);
    }
  }
}

// greedily keep the max detections (already sorted)
inline void NonMaximumSuppression(float* dets,
                                  int post_nms_top_n,
                                  int num_images,
                                  int num_anchors,
                                  int width,
                                  int height,
                                  std::vector< std::vector<int> > & final_keep_images,
                                  std::vector< std::vector<int> > & suppression_list,
                                  std::vector<float> anchors,
                                  float feature_stride,
                                  float thresh) {

  int total_anchors = num_images*num_anchors*width*height;
  int chip_anchors = num_anchors*width*height;

  float *area = new float[total_anchors];

  #pragma omp parallel for num_threads(8)
  for (int i = 0; i < total_anchors; ++i) {
    area[i] = (dets[5*i + 2] - dets[5*i + 0] + 1) * (dets[5*i + 3] - dets[5*i + 1] + 1);
  }

  int max_nms = std::min(12000, chip_anchors);

  #pragma omp parallel for num_threads(8)
  for (int i = 0; i < num_images; i++) {
    std::vector <float> sortids(chip_anchors);
    for (int j = 0; j < chip_anchors; j++) {
      sortids[j] = j;
    }
    int chip_index = i*chip_anchors;

    std::sort(sortids.begin(), sortids.end(),
        [&dets,chip_index](int i1, int i2) {
          return dets[5*(chip_index + i1) + 4] > dets[5*(chip_index + i2) + 4];
        });

    float *dbuf = new float[6*max_nms];

    //reorder for spatial locality in CPU, yo!
    for (int j = 0; j < max_nms; j++) {
      int index = i*chip_anchors + sortids[j];
      dbuf[6*j] = dets[5*index];
      dbuf[6*j+1] = dets[5*index+1];
      dbuf[6*j+2] = dets[5*index+2];
      dbuf[6*j+3] = dets[5*index+3];
      dbuf[6*j+4] = dets[5*index+4];
      dbuf[6*j+5] = area[index];
    }

    int vct = 0;

    int j = 0;
    for (; j < max_nms && vct < post_nms_top_n; j++) {
      int index = i*chip_anchors + sortids[j];
      float ix1 = dbuf[6*j];
      float iy1 = dbuf[6*j+1];
      float ix2 = dbuf[6*j+2];
      float iy2 = dbuf[6*j+3];
      float iarea = dbuf[6*j+5];

      if (dbuf[6*j+4] == -1) {
        continue;
      }

      final_keep_images[i].push_back(index);
      vct = vct + 1;
      for (int pind = j + 1; pind < max_nms; pind++) {
        if (dbuf[6*pind + 4] == -1) {
          continue;
        }
        float xx1 = std::max(ix1, dbuf[6*pind]);
        float yy1 = std::max(iy1, dbuf[6*pind + 1]);
        float xx2 = std::min(ix2, dbuf[6*pind + 2]);
        float yy2 = std::min(iy2, dbuf[6*pind + 3]);
        float w = std::max(0.0f, xx2 - xx1 + 1.0f);
        float h = std::max(0.0f, yy2 - yy1 + 1.0f);
        float inter = w * h;
        float ovr = inter / (iarea + dbuf[6*pind+5] - inter);
        if (ovr > 0.7) {
          dbuf[6*pind + 4] = -1;
        }
      }
    }
    delete [] dbuf;
  }
  delete [] area;
}

// For each index, check if its in the top max_nms, if it is calculate the iou, and set score to -1, if iou > 0.7.
  inline void compute_iou(int ind, float* dets, float* area, int max_nms, float ix1, float iy1, float ix2, float iy2, float iarea){
        if( dets[ind*5 + 4] != -1){
            float xx1 = std::max(ix1, dets[5*ind]);
            float yy1 = std::max(iy1, dets[5*ind + 1]);
            float xx2 = std::min(ix2, dets[5*ind + 2]);
            float yy2 = std::min(iy2, dets[5*ind + 3]);
            float w = std::max(0.0f, xx2 - xx1 + 1.0f);
            float h = std::max(0.0f, yy2 - yy1 + 1.0f);
            float inter = w * h;
            float ovr = inter / (iarea + area[ind] - inter);
            if (ovr > 0.7) {
              dets[5*ind + 4] = -1;
            }
          }

  }


inline void FastNonMaxSuppression(float* dets,
                                  std::vector<std::vector<int>> & anchor_iou_map,
                                  int post_nms_top_n,
                                  int num_images,
                                  int num_anchors,
                                  int width,
                                  int height,
                                  std::vector<std::vector<int>> & final_keep_images,
                                  std::vector<std::vector<int>> & suppression_list){

  int total_anchors = num_images*num_anchors*width*height;
  int chip_anchors = num_anchors*width*height;

  float* area = new float[total_anchors];

  // Pre compute area.
  #pragma omp parallel for num_threads(8)
  for (int i = 0; i < total_anchors; ++i) {
    area[i] = (dets[5*i + 2] - dets[5*i + 0] + 1) * (dets[5*i + 3] - dets[5*i + 1] + 1);
  }

  int max_nms = std::min(12000, chip_anchors);

  #pragma omp parallel for num_threads(8)
  for (int i = 0; i < num_images; i++) {
    std::vector <float> sortids(chip_anchors);
    for (int j = 0; j < chip_anchors; j++) {
      sortids[j] = j;
    }
    int chip_index = i*chip_anchors;
    auto start_sort = std::chrono::high_resolution_clock::now();
    std::nth_element(sortids.begin(), sortids.begin()+max_nms, sortids.end(),
        [&dets,chip_index](int i1, int i2) {
          return dets[5*(chip_index + i1) + 4] > dets[5*(chip_index + i2) + 4];
        });
    std::sort(sortids.begin(), sortids.begin()+max_nms,
        [&dets,chip_index](int i1, int i2) {
          return dets[5*(chip_index + i1) + 4] > dets[5*(chip_index + i2) + 4];
        });
 

    auto stop_sort = std::chrono::high_resolution_clock::now();
    auto duration_sort = std::chrono::duration_cast<std::chrono::microseconds>(stop_sort - start_sort);

    float *dbuf = new float[6*max_nms];

    //reorder for spatial locality in CPU, yo!
    // reorder indexes, for top max_nms scoring, proposals.
    for (int j = 0; j < max_nms; j++) {
      int index = i*chip_anchors + sortids[j];
      dbuf[6*j] = dets[5*index];
      dbuf[6*j+1] = dets[5*index+1];
      dbuf[6*j+2] = dets[5*index+2];
      dbuf[6*j+3] = dets[5*index+3];
      dbuf[6*j+4] = dets[5*index+4];
      dbuf[6*j+5] = area[index];
    }

    int vct = 0;
    int im_area = width * height;
    int idx_1;
    int w_1, h_1, c_1, w_2, h_2, c_2;
    int rel_w, rel_h;
//    int iarea, pind;
    std::vector<int> compute_idx;
    int chip_range = i*chip_anchors;
    // retrieve a proposal from the top ranked scoring proposals.
    int j = 0;
    auto start_loop = std::chrono::high_resolution_clock::now();
    for (; j < max_nms && vct < post_nms_top_n; j++) {
      int index = chip_range + sortids[j];
      float ix1 = dbuf[6*j];
      float iy1 = dbuf[6*j+1];
      float ix2 = dbuf[6*j+2];
      float iy2 = dbuf[6*j+3];
      float iarea = dbuf[6*j+5];

      idx_1 = sortids[j];
      int mat = idx_1 % (im_area);
      w_1 = mat % width;
      h_1 = mat / width;
      c_1 = idx_1 / im_area;

      if (dbuf[6*j+4] == -1 || dets[5*index + 4] == -1) {
        continue;
      }

      idx_1 = sortids[j];

      final_keep_images[i].push_back(index);
      vct = vct + 1;

      // retrieve the precomputed anchor overlap vector for the given proposal
      // each entry in the anchor_iou_map consists of a tuple containing relative width, relative height
      // and channel of the anchor, with which the overlap of current proposal's anchor was higher than certain
      // threshold.

      compute_idx = anchor_iou_map[c_1];
      int pind;
      for(int i = 0; i < compute_idx.size()/3; i++){
      // retrieve each entry of relative width, height and anchor channel.
        rel_w = compute_idx[i*3+0];
        rel_h = compute_idx[i*3+1];
        c_2   = compute_idx[i*3+2];
        w_2   = w_1 + rel_w;
        h_2   = h_1 + rel_h;
        pind  = chip_range + c_2*(height*width) + h_2 * width + w_2;

        if(w_2 < width && w_2 >=0 && h_2 < height && h_2 >=0){
            compute_iou(pind, dets, area, max_nms, ix1, iy1, ix2, iy2, iarea);
        }

      }
    }

    auto stop_loop = std::chrono::high_resolution_clock::now();
    auto duration_loop = std::chrono::duration_cast<std::chrono::microseconds>(stop_loop - start_loop);
    delete [] dbuf;
  }
  delete [] area;
  }

  inline void insert_anchor_iou(std::vector<std::vector<int>> &anchor_iou_dp, int anchor_type_1, int anchor_type_2, int dx, int dy){
    // anchor 1 intersection with anchor 2
    anchor_iou_dp[anchor_type_1].push_back(dx);
    anchor_iou_dp[anchor_type_1].push_back(dy);
    anchor_iou_dp[anchor_type_1].push_back(anchor_type_2);
  }

}  // namespace utils


template<typename xpu>
class MultiProposalGPUOp : public Operator{
 public:
 	float *scores;
 	float *bbox_deltas;
 	float *proposals;
 	float *im_info;
 	float *rois;
  float *out_scores;
     float* anchor_area;
      std::vector<std::vector<int>> anchor_iou_dp;
    bool is_anchor_overlap_precomputed;



  explicit MultiProposalGPUOp(MultiProposalParam param) {
    this->param_ = param;
    int batch_size = param.batch_size;    
    this->scores = new float[batch_size*21*2*200*200];
    this->bbox_deltas = new float[batch_size*21*4*200*200];
    this->proposals = new float[batch_size*21*5*200*200];
    this->im_info = new float[batch_size*3];
    this->rois = new float[param.rpn_post_nms_top_n * batch_size * 5];
    this->out_scores = new float[param.rpn_post_nms_top_n*batch_size];
    this->anchor_area = new float[21];
    this->is_anchor_overlap_precomputed = false;

  }

  ~MultiProposalGPUOp() {
    delete [] this->scores;
    delete [] this->bbox_deltas;
    delete [] this->proposals;
    delete [] this->im_info;
    delete [] this->rois;
    delete [] this->out_scores;
  }

  virtual void Forward(const OpContext &ctx,
                       const std::vector<TBlob> &in_data,
                       const std::vector<OpReqType> &req,
                       const std::vector<TBlob> &out_data,
                       const std::vector<TBlob> &aux_states) {
    bool useFastNMS = true;
    
    CHECK_EQ(in_data.size(), 3);
    CHECK_EQ(out_data.size(), 2);
    
    using namespace mshadow;
    using namespace mshadow::expr;
    //clock_t t;
  	//t = clock();
    Stream<gpu> *s = ctx.get_stream<gpu>();
    Tensor<gpu, 4> tscores = in_data[proposal::kClsProb].get<gpu, 4, real_t>(s);
    Tensor<gpu, 4> tbbox_deltas = in_data[proposal::kBBoxPred].get<gpu, 4, real_t>(s);
    Tensor<gpu, 2> tim_info = in_data[proposal::kImInfo].get<gpu, 2, real_t>(s);

    int num_images = tbbox_deltas.size(0);
    int num_anchors = tbbox_deltas.size(1) / 4;
    int height = tbbox_deltas.size(2);
    int width = tbbox_deltas.size(3);
    int count_anchors = num_anchors*height*width;
    int total_anchors = count_anchors * num_images;

    hipMemcpy(scores, tscores.dptr_, total_anchors*2*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(bbox_deltas, tbbox_deltas.dptr_, total_anchors*4*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(im_info, tim_info.dptr_, 3 * sizeof(float) * num_images, hipMemcpyDeviceToHost);
    

    std::vector<float> base_anchor(4);
    //usleep(20000000);
    base_anchor[0] = 0.0;
    base_anchor[1] = 0.0;
    base_anchor[2] = param_.feature_stride - 1.0;
    base_anchor[3] = param_.feature_stride - 1.0;

    std::vector<float> anchors;
    utils::GenerateAnchors(base_anchor,
                           param_.ratios,
                           param_.scales,
                           &anchors);

    float a1_x1, a1_x2, a1_y1, a1_y2, a2_x1, a2_x2, a2_y1, a2_y2, a1_area, a2_area;
//    #pragma omp parallel for num_threads(8)
    for(int i = 0; i < num_anchors; i++){
        a1_x1 = anchors[4*i + 0];
        a1_y1 = anchors[4*i + 1];
        a1_x2 = anchors[4*i + 2];
        a1_y2 = anchors[4*i + 3];
        a1_area = (a1_x2 - a1_x1 + 1)*(a1_y2 - a1_y1 + 1);
        anchor_area[i] = a1_area;
    }

    //std::cout << "quack 3" << std::endl;
    #pragma omp parallel for num_threads(8)
    for (int t = 0; t < total_anchors; ++t) {
      int b = t / count_anchors;
      int index = t % count_anchors;
      int i = index / (height*width);
      int mat = t % (height*width);
      int k = mat % width; //width index
      int j = mat / width; //height index
      proposals[5*t] = anchors[4*i] + k * param_.feature_stride;
      proposals[5*t + 1] = anchors[4*i+1] + j * param_.feature_stride;
      proposals[5*t + 2] = anchors[4*i+2] + k * param_.feature_stride;
      proposals[5*t + 3] = anchors[4*i+3] + j * param_.feature_stride;
          // suppress anchors of type t1 for stride x%,
      if(((i+4)%7==0 || (i+2)%7==0)){
        proposals[5*t + 4] = -1;
        continue;
      }
      proposals[5*t + 4] = scores[b*count_anchors*2 + ((num_anchors + i)*height + j)*width + k];
    }

    std::vector<std::vector<int>> suppression_list(num_anchors);

    for(int i = 0; i < num_anchors; i++){
        suppression_list[i] = std::vector<int>(0);
    }
    // Precompute the anchor overlap
    if (!is_anchor_overlap_precomputed){
        for(int i = 0; i < num_anchors; i++){
            anchor_iou_dp.push_back(std::vector<int>());
        }
        
        float thresh = param_.roi_iou_thresh;
        int a1_c, a2_c;
        int feat_size = width*height;

        for(int i = 0; i < num_anchors; i++){
            a1_c = i;
            a1_x1 = anchors[4*i + 0];
            a1_y1 = anchors[4*i + 1];
            a1_x2 = anchors[4*i + 2];
            a1_y2 = anchors[4*i + 3];
            a1_area = anchor_area[i];
            for(int t = 0; t < count_anchors; t++){

              a2_c = t / (feat_size);

              int mat = t % (feat_size);
              int k = mat % width; //width index
              int j = mat / width; //height index

              a2_area = anchor_area[a2_c];

              a2_x1 = proposals[5*t+0];
              a2_y1 = proposals[5*t+1];
              a2_x2 = proposals[5*t+2];
              a2_y2 = proposals[5*t+3];
               //Compute iou between anchor 1 & anchor 2
              float xx1 = std::max(a1_x1, a2_x1);
              float yy1 = std::max(a1_y1, a2_y1);
              float xx2 = std::min(a1_x2, a2_x2);
              float yy2 = std::min(a1_y2, a2_y2);
              float w = std::max(0.0f, xx2 - xx1 + 1.0f);
              float h = std::max(0.0f, yy2 - yy1 + 1.0f);
              float inter = w * h;
              float ovr = inter / (a1_area + a2_area - inter);

              if(j==0 && k==0 && a2_c==a1_c){
                    continue;
              }


              if (ovr >= thresh){
              // +dx +dy
                utils::insert_anchor_iou(anchor_iou_dp, a1_c, a2_c, j, k);
                utils::insert_anchor_iou(anchor_iou_dp, a1_c, a2_c, -j, k);
                utils::insert_anchor_iou(anchor_iou_dp, a1_c, a2_c, -j, -k);
                utils::insert_anchor_iou(anchor_iou_dp, a1_c, a2_c, j, -k);
              }

            }
        }
        is_anchor_overlap_precomputed = true;
    }

    utils::BBoxTransformInv(proposals, bbox_deltas, im_info, num_images, num_anchors, height, width);

    utils::FilterBox(proposals, total_anchors, 3);

    std::vector <std::vector<int> > keep_images(num_images);
    for (int i = 0; i < num_images; i++) {
      keep_images[i] = std::vector<int>(0);
    }
    int rpn_post_nms_top_n = param_.rpn_post_nms_top_n;

    auto start = std::chrono::high_resolution_clock::now();


    if (useFastNMS) {
      utils::FastNonMaxSuppression(proposals, anchor_iou_dp, rpn_post_nms_top_n, num_images, num_anchors, width, height,
                                    keep_images, suppression_list);
    }
    else {
      utils::NonMaximumSuppression(proposals, rpn_post_nms_top_n, num_images, num_anchors, width, height, keep_images,
                                    suppression_list, anchors, param_.feature_stride, param_.roi_iou_thresh);
    }

    auto stop = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
    
    #pragma omp parallel for num_threads(8)
    for (int i = 0; i < num_images; i++) {
      int numpropsi = keep_images[i].size();
      for (int j = 0; j < numpropsi; j++) {
        int base = (i*rpn_post_nms_top_n + j);
        rois[5*base] = i;
        rois[5*base+1] = proposals[5*keep_images[i][j] + 0];
        rois[5*base+2] = proposals[5*keep_images[i][j] + 1];
        rois[5*base+3] = proposals[5*keep_images[i][j] + 2];
        rois[5*base+4] = proposals[5*keep_images[i][j] + 3];
        out_scores[base] = proposals[5*keep_images[i][j] + 4];
      }

      for (int j = numpropsi; j < rpn_post_nms_top_n; j++) {
        int base = (i*rpn_post_nms_top_n + j);
        rois[5*base+0] = i;
        rois[5*base+1] = rand() % 100;
        rois[5*base+2] = rand() % 100;
        rois[5*base+3] = 200 + rand() % 200;
        rois[5*base+4] = 200 + rand() % 200;
        out_scores[base] = 0.0;
      }
    }

    Stream<gpu> *so = ctx.get_stream<gpu>();
    Tensor<gpu,1> oscores = out_data[proposal::kScores].get<gpu, 1, real_t>(so);    
    Tensor<gpu, 2> orois = out_data[proposal::kRoIs].get<gpu, 2, real_t>(so);
    hipMemcpy(orois.dptr_, rois, 5*sizeof(float) * num_images * rpn_post_nms_top_n, hipMemcpyHostToDevice);
    hipMemcpy(oscores.dptr_, out_scores, sizeof(float) * num_images * rpn_post_nms_top_n, hipMemcpyHostToDevice);
  }

  virtual void Backward(const OpContext &ctx,
                        const std::vector<TBlob> &out_grad,
                        const std::vector<TBlob> &in_data,
                        const std::vector<TBlob> &out_data,
                        const std::vector<OpReqType> &req,
                        const std::vector<TBlob> &in_grad,
                        const std::vector<TBlob> &aux_states) {
    using namespace mshadow;
    using namespace mshadow::expr;
    CHECK_EQ(in_grad.size(), 4);

    Stream<xpu> *s = ctx.get_stream<xpu>();
    Tensor<xpu, 4> gscores = in_grad[proposal::kClsProb].get<xpu, 4, real_t>(s);
    Tensor<xpu, 4> gbbox = in_grad[proposal::kBBoxPred].get<xpu, 4, real_t>(s);
    Tensor<xpu, 2> ginfo = in_grad[proposal::kImInfo].get<xpu, 2, real_t>(s);

    // can not assume the grad would be zero
    Assign(gscores, req[proposal::kClsProb], 0);
    Assign(gbbox, req[proposal::kBBoxPred], 0);
    Assign(ginfo, req[proposal::kImInfo], 0);
  }

 private:
  MultiProposalParam param_;
};  // class MultiProposalOp

template<>
Operator *CreateOp<gpu>(MultiProposalParam param) {
  return new MultiProposalGPUOp<gpu>(param);
}


}  // namespace op
}  // namespace mxnet
